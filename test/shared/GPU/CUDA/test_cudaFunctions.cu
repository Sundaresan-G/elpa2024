//
//    Copyright 2014, A. Marek
//
//    This file is part of ELPA.
//
//    The ELPA library was originally created by the ELPA consortium,
//    consisting of the following organizations:
//
//    - Max Planck Computing and Data Facility (MPCDF), formerly known as
//      Rechenzentrum Garching der Max-Planck-Gesellschaft (RZG),
//    - Bergische Universität Wuppertal, Lehrstuhl für angewandte
//      Informatik,
//    - Technische Universität München, Lehrstuhl für Informatik mit
//      Schwerpunkt Wissenschaftliches Rechnen ,
//    - Fritz-Haber-Institut, Berlin, Abt. Theorie,
//    - Max-Plack-Institut für Mathematik in den Naturwissenschaften,
//      Leipzig, Abt. Komplexe Strukutren in Biologie und Kognition,
//      and
//    - IBM Deutschland GmbH
//
//    This particular source code file contains additions, changes and
//    enhancements authored by Intel Corporation which is not part of
//    the ELPA consortium.
//
//    More information can be found here:
//    http://elpa.mpcdf.mpg.de/
//
//    ELPA is free software: you can redistribute it and/or modify
//    it under the terms of the version 3 of the license of the
//    GNU Lesser General Public License as published by the Free
//    Software Foundation.
//
//    ELPA is distributed in the hope that it will be useful,
//    but WITHOUT ANY WARRANTY; without even the implied warranty of
//    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//    GNU Lesser General Public License for more details.
//
//    You should have received a copy of the GNU Lesser General Public License
//    along with ELPA.  If not, see <http://www.gnu.org/licenses/>
//
//    ELPA reflects a substantial effort on the part of the original
//    ELPA consortium, and we ask you to respect the spirit of the
//    license that we chose: i.e., please contribute any changes you
//    may have back to the original ELPA library distribution, and keep
//    any derivatives of ELPA under the same license that we chose for
//    the original distribution, the GNU Lesser General Public License.
//
//
// --------------------------------------------------------------------------------------------------
//
// This file was written by A. Marek, MPCDF
#include "config-f90.h"

#include <stdio.h>
#include <math.h>
#include <stdio.h>

#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <alloca.h>
#include <stdint.h>
#include <complex.h>
#ifdef WITH_NVIDIA_GPU_VERSION
#include <hipblas.h>
#endif

#include "./test_cudaFunctions.h"

#define errormessage(x, ...) do { fprintf(stderr, "%s:%d " x, __FILE__, __LINE__, __VA_ARGS__ ); } while (0)

#ifdef DEBUG_CUDA
#define debugmessage(x, ...) do { fprintf(stderr, "%s:%d " x, __FILE__, __LINE__, __VA_ARGS__ ); } while (0)
#else
#define debugmessage(x, ...)
#endif

#ifdef WITH_NVIDIA_GPU_VERSION
extern "C"
{
  int cudaSetDeviceFromC(int n) {

    hipError_t cuerr = hipSetDevice(n);
    if (cuerr != hipSuccess) {
      errormessage("Error in hipSetDevice(%i): %s\n", n, hipGetErrorString(cuerr));
      return 0;
    }
    return 1;
  }

  int cudaGetDeviceCountFromC(int *count) {

    hipError_t cuerr = hipGetDeviceCount(count);
    if (cuerr != hipSuccess) {
      errormessage("Error in hipGetDeviceCount: %s\n",hipGetErrorString(cuerr));
      return 0;
    }
    return 1;
  }
  
  int cudaMallocFromC(intptr_t *a, size_t width_height) {

    hipError_t cuerr = hipMalloc((void **) a, width_height);
#ifdef DEBUG_CUDA
    printf("CUDA Malloc,  pointer address: %p, size: %d \n", *a, width_height);
#endif
    if (cuerr != hipSuccess) {
      errormessage("Error in hipMalloc: %s\n",hipGetErrorString(cuerr));
      return 0;
    }
    return 1;
  }
  
  int cudaFreeFromC(intptr_t *a) {
#ifdef DEBUG_CUDA
    printf("CUDA Free, pointer address: %p \n", a);
#endif
    hipError_t cuerr = hipFree(a);

    if (cuerr != hipSuccess) {
      errormessage("Error in hipFree: %s\n",hipGetErrorString(cuerr));
      return 0;
    }
    return 1;
  }

  int cudaMemcpyFromC(intptr_t *dest, intptr_t *src, size_t count, int dir) {

    hipError_t cuerr = hipMemcpy( dest, src, count, (hipMemcpyKind)dir);
    if (cuerr != hipSuccess) {
      errormessage("Error in hipMemcpy: %s\n",hipGetErrorString(cuerr));
      return 0;
    }
    return 1;
  }

  int cudaMemcpyDeviceToDeviceFromC(void) {
      int val = hipMemcpyDeviceToDevice;
      return val;
  }
  int cudaMemcpyHostToDeviceFromC(void) {
      int val = hipMemcpyHostToDevice;
      return val;
  }
  int cudaMemcpyDeviceToHostFromC(void) {
      int val = hipMemcpyDeviceToHost;
      return val;
  }
  int cudaHostRegisterDefaultFromC(void) {
      int val = hipHostRegisterDefault;
      return val;
  }
  int cudaHostRegisterPortableFromC(void) {
      int val = hipHostRegisterPortable;
      return val;
  }
  int cudaHostRegisterMappedFromC(void) {
      int val = hipHostRegisterMapped;
      return val;
  }
}
#endif /* TEST_NVIDIA_GPU == 1 */
