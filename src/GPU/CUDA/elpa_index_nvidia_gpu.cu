
#include <hip/hip_runtime.h>
extern "C" {
  int nvidia_gpu_count() {
    int count;
    hipError_t cuerr = hipGetDeviceCount(&count);
    if (cuerr != hipSuccess) {
      count = -1000;
    }
    return count;
  }
}
