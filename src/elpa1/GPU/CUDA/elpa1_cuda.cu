#include "hip/hip_runtime.h"
//    Copyright 2023, P. Karpov
//
//    This file is part of ELPA.
//
//    The ELPA library was originally created by the ELPA consortium,
//    consisting of the following organizations:
//
//    - Max Planck Computing and Data Facility (MPCDF), formerly known as
//      Rechenzentrum Garching der Max-Planck-Gesellschaft (RZG),
//    - Bergische Universität Wuppertal, Lehrstuhl für angewandte
//      Informatik,
//    - Technische Universität München, Lehrstuhl für Informatik mit
//      Schwerpunkt Wissenschaftliches Rechnen ,
//    - Fritz-Haber-Institut, Berlin, Abt. Theorie,
//    - Max-Plack-Institut für Mathematik in den Naturwissenschaften,
//      Leipzig, Abt. Komplexe Strukutren in Biologie und Kognition,
//      and
//    - IBM Deutschland GmbH
//
//    This particular source code file contains additions, changes and
//    enhancements authored by Intel Corporation which is not part of
//    the ELPA consortium.
//
//    More information can be found here:
//    http://elpa.mpcdf.mpg.de/
//
//    ELPA is free software: you can redistribute it and/or modify
//    it under the terms of the version 3 of the license of the
//    GNU Lesser General Public License as published by the Free
//    Software Foundation.
//
//    ELPA is distributed in the hope that it will be useful,
//    but WITHOUT ANY WARRANTY; without even the implied warranty of
//    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//    GNU Lesser General Public License for more details.
//
//    You should have received a copy of the GNU Lesser General Public License
//    along with ELPA.  If not, see <http://www.gnu.org/licenses/>
//
//    ELPA reflects a substantial effort on the part of the original
//    ELPA consortium, and we ask you to respect the spirit of the
//    license that we chose: i.e., please contribute any changes you
//    may have back to the original ELPA library distribution, and keep
//    any derivatives of ELPA under the same license that we chose for
//    the original distribution, the GNU Lesser General Public License.
//
//    This file was written by P. Karpov, MPCDF

#include <stdio.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <alloca.h>
#include <complex.h>
#include <hip/hip_complex.h>
#include <stdint.h>
#include "config-f90.h"

#define errormessage(x, ...) do { fprintf(stderr, "%s:%d " x, __FILE__, __LINE__, __VA_ARGS__ ); } while (0)

/*
template <typename T>
void sycl_copy_a_tmat2_kernel(T *a_dev, T *tmat2_dev, const int nblk, const int matrixRows, const int l_colx, const int l_row1, sycl::nd_item<1> it){

  int nb_index = it.get_local_id(0) + 1; // range 1..nb
  int l_col_index = it.get_group(0) + 1; // range 1..l_colx-l_cols-1

  tmat2_dev[nb_index-1 + (l_colx-1 + l_col_index -1) * nblk] = a_dev[l_row1-1 + nb_index-1 + (l_colx-1 + l_col_index -1)  * matrixRows];

}

template <typename T>
void sycl_copy_a_tmat2_FromC(T *a_dev, T *tmat2_dev, int *nblk_in, int *matrixRows_in, int *l_cols_in, int *l_colx_in, int *l_row1_in, int *nb_in, intptr_t my_stream){

  int nblk = *nblk_in;   
  int matrixRows = *matrixRows_in;
  int l_cols = *l_cols_in;
  int l_colx = *l_colx_in;
  int l_row1 = *l_row1_in;
  int nb     = *nb_in;

  sycl::range<1> global_range = sycl::range<1>(nb*(l_cols - l_colx + 1));
  sycl::range<1> local_range  = sycl::range<1>(nb);

  auto device = elpa::gpu::sycl::getDevice();
  auto &queue = elpa::gpu::sycl::getQueue();

  queue.parallel_for(
      sycl::nd_range<1>(global_range, local_range),
      [=](sycl::nd_item<1> it) {
        sycl_copy_a_tmat2_kernel(a_dev, tmat2_dev, nblk, matrixRows,
                                        l_colx, l_row1, it);
      });
  queue.wait_and_throw();

}

extern "C" void sycl_copy_double_a_tmat2_FromC(double *a_dev, double *tmat2_dev, int *nblk_in, int *matrixRows_in, int *l_cols_in, int *l_colx_in, int *l_row1_in, int *nb_in, intptr_t my_stream){
  sycl_copy_a_tmat2_FromC(a_dev, tmat2_dev, nblk_in, matrixRows_in, l_cols_in, l_colx_in, l_row1_in, nb_in, my_stream);
}

extern "C" void sycl_copy_float_a_tmat2_FromC(float *a_dev, float *tmat2_dev, int *nblk_in, int *matrixRows_in, int *l_cols_in, int *l_colx_in, int *l_row1_in, int *nb_in, intptr_t my_stream){
  sycl_copy_a_tmat2_FromC(a_dev, tmat2_dev, nblk_in, matrixRows_in, l_cols_in, l_colx_in, l_row1_in, nb_in, my_stream);
}

extern "C" void sycl_copy_double_complex_a_tmat2_FromC(std::complex<double> *a_dev, std::complex<double> *tmat2_dev, int *nblk_in, int *matrixRows_in, int *l_cols_in, int *l_colx_in, int *l_row1_in, int *nb_in, intptr_t my_stream){
  sycl_copy_a_tmat2_FromC(a_dev, tmat2_dev, nblk_in, matrixRows_in, l_cols_in, l_colx_in, l_row1_in, nb_in, my_stream);
}

extern "C" void sycl_copy_float_complex_a_tmat2_FromC(std::complex<float> *a_dev, std::complex<float> *tmat2_dev, int *nblk_in, int *matrixRows_in, int *l_cols_in, int *l_colx_in, int *l_row1_in, int *nb_in, intptr_t my_stream){
  sycl_copy_a_tmat2_FromC(a_dev, tmat2_dev, nblk_in, matrixRows_in, l_cols_in, l_colx_in, l_row1_in, nb_in, my_stream);
}
*/
//________________________________________________________________

__global__ void cuda_update_matrix_element_add_double_kernel(double *a_dev, int index, double value, double *d_vec_dev, int istep, int n_stored_vecs, int const isSkewsymmetric){
  if (n_stored_vecs > 0){
    a_dev[index] += value;
    }

    if (isSkewsymmetric) {
      d_vec_dev[istep-1-1] = 0.0;
    }
    else {
      d_vec_dev[istep-1-1] = a_dev[index]; // (l_rows,l_cols)
    }
/*
#endif
#if COMPLEXCASE == 1
      d_vec(istep-1) = real(a_mat(l_rows,l_cols),kind=rk)
#endif
*/
}

extern "C" void cuda_update_matrix_element_add_double_FromC(double *a_dev, int *index_in, double *value_in, 
                          double *d_vec_dev, int *istep_in, int *n_stored_vecs_in, int* isSkewsymmetric_in, hipStream_t  my_stream){
  int index = *index_in;   
  int istep = *istep_in;   
  int n_stored_vecs = *n_stored_vecs_in; 
  int isSkewsymmetric = *isSkewsymmetric_in;   
  double value = *value_in;

  dim3 blocks = dim3(1,1,1);
  dim3 threadsPerBlock = dim3(1,1,1);

#ifdef WITH_GPU_STREAMS
  cuda_update_matrix_element_add_double_kernel<<<blocks,threadsPerBlock,0,my_stream>>>(a_dev, index, value, d_vec_dev, istep, n_stored_vecs, isSkewsymmetric);
#else
  cuda_update_matrix_element_add_double_kernel<<<blocks,threadsPerBlock>>>(a_dev, index, value, d_vec_dev, istep, n_stored_vecs, isSkewsymmetric);
#endif
  hipError_t cuerr = hipGetLastError();
  if (cuerr != hipSuccess){
    printf("Error in executing cuda_update_matrix_element_add_double_kernel: %s\n",hipGetErrorString(cuerr));
  }
}

//________________________________________________________________

__global__ void cuda_update_array_element_double_kernel(double *array_dev, const int index, double value){

  array_dev[index-1] = value;

}

extern "C" void cuda_update_array_element_double_FromC(double *array_dev, int *index_in, double *value_in, hipStream_t  my_stream){
  int index = *index_in;   
  double value = *value_in;

  dim3 blocks = dim3(1,1,1);
  dim3 threadsPerBlock = dim3(1,1,1);

#ifdef WITH_GPU_STREAMS
  cuda_update_array_element_double_kernel<<<blocks,threadsPerBlock,0,my_stream>>>(array_dev, index, value);
#else
  cuda_update_array_element_double_kernel<<<blocks,threadsPerBlock>>>(array_dev, index, value);
#endif
  hipError_t cuerr = hipGetLastError();
  if (cuerr != hipSuccess){
    printf("Error in executing cuda_update_array_element_double_kernel: %s\n",hipGetErrorString(cuerr));
  }
}
